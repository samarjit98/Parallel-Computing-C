
#include <hip/hip_runtime.h>
#include "stdio.h"

__global__ void add(int *a, int *b, int *c, int *N){
	 int tID = blockIdx.x;
	 if (tID < *N){
	 	c[tID] = a[tID] + b[tID];
	 }
}

int main(int argc, char* argv[]){

	int N = atoi(argv[1]);
	int *a, *b, *c, *n;
	int *dev_a, *dev_b, *dev_c, *dev_n;

	 a = (int*)malloc(N*sizeof(int));
	 b = (int*)malloc(N*sizeof(int));
	 c = (int*)malloc(N*sizeof(int));
	 n = (int*)malloc(sizeof(int));

	 hipMalloc((void **) &dev_a, N*sizeof(int));
	 hipMalloc((void **) &dev_b, N*sizeof(int));
	 hipMalloc((void **) &dev_c, N*sizeof(int));
	 hipMalloc((void **) &dev_n, sizeof(int));
	 // Fill Arrays
	 for (int i = 0; i < N; i++)
	 {
		 a[i] = rand()%10000;
		 b[i] = rand()%10000;
	 }
	 *n = N;

	 hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	 hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
	 hipMemcpy(dev_n, n, sizeof(int), hipMemcpyHostToDevice);

	 add<<<N,1>>>(dev_a, dev_b, dev_c, dev_n);

	 hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

	 for (int i = 0; i < N; i++){
	 	printf("%d + %d = %d\n", a[i], b[i], c[i]);
	 }

	 return 0;
}